#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "NeuralNetwork/CUDAMath.cuh"
#if USE_CUBLAS
#include <hipblas.h>
#endif

// Kernels

// A: aRows x aCols -
// B: aCols x bCols - x m
// C: aRows x bCols -
__global__ void MatMulKernel(const float* A, const float* B, float* C, int aRows, int aCols, int bCols)
{
	const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col < bCols && row < aRows)
	{
        float sum = 0;
		for (int i = 0; i < aCols; i++)
		{
			sum += A[row * aCols + i] * B[i * bCols + col];
		}
		C[row * bCols + col] = sum;
	}
}

__global__ void MatAddInKernel(float* A, float* B, unsigned int maxNum)
{
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		A[i] += B[i];
}

__global__ void MatAddKerlen(float* A, float* B, float* C, unsigned int maxNum)
{
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		C[i] = A[i] + B[i];
}

__global__ void MatSubInKernel(float* A, float* B, unsigned int maxNum)
{
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		A[i] -= B[i];
}

__global__ void MatSubKernel(float* A, float* B, float* C, unsigned int maxNum)
{
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		C[i] = A[i] - B[i];
}

__global__ void InnerProductKernel(float* A, float* B, unsigned int maxNum)
{
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		A[i] *= B[i];
}

__global__ void FillKernel(float* a, float val, unsigned int maxNum)
{
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		a[i] = val;
}

__global__ void AddConstKernel(float* A, float v, unsigned int maxNum)
{
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		A[i] += v;
}

__global__ void SubConstKernel(float* A, float v, unsigned int maxNum)
{
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		A[i] -= v;
}

__global__ void MulConstKernel(float* A, float v, unsigned int maxNum)
{
	const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		A[i] *= v;
}

// Addition
void MatrixCUDAMath::Add(const Matrix& a, const Matrix& b, Matrix& c)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	MatAddKerlen <<<grid, threads >>> (a.GetConstGPUValues(), b.GetConstGPUValues(), c.GetGPUValues(), max);
}


void MatrixCUDAMath::AddIn(Matrix& a, const Matrix& b)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	MatAddInKernel <<<grid, threads >>> (a.GetGPUValues(), b.GetConstGPUValues(), max);
}

void MatrixCUDAMath::AddConstant(Matrix& a, float v)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	AddConstKernel <<<grid, threads >>> (a.GetGPUValues(), v, max);
}

// Subtraction
void MatrixCUDAMath::Subtract(const Matrix& a, const Matrix& b, Matrix& c)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	MatSubKernel <<<grid, threads >>> (a.GetConstGPUValues(), b.GetConstGPUValues(), c.GetGPUValues(), max);
}

void MatrixCUDAMath::SubtractIn(Matrix& a, const Matrix& b)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	MatSubInKernel <<<grid, threads >>> (a.GetGPUValues(), b.GetConstGPUValues(), max);
}

void MatrixCUDAMath::SubtractConstant(Matrix& a, float v)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	SubConstKernel <<<grid, threads >>> (a.GetGPUValues(), v, max);
}

//Multiplication
void MatrixCUDAMath::Multiplication(const Matrix& a, const Matrix& b, Matrix& c)
{
	unsigned int rows = ceil((double)(a.GetRowCount() + BLOCK_SIZE - 1) / (double)BLOCK_SIZE);
	unsigned int cols = ceil((double)(b.GetColumnCount() + BLOCK_SIZE - 1) / (double)BLOCK_SIZE);

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(cols, rows);
	MatMulKernel <<<blocks, threads >>> (a.GetConstGPUValues(), b.GetConstGPUValues(), c.GetGPUValues(), a.GetRowCount(), a.GetColumnCount(), b.GetColumnCount());
}

void MatrixCUDAMath::ElementwiseMultiply(Matrix& a, const Matrix& b)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	InnerProductKernel <<<grid, threads >>> (a.GetGPUValues(), b.GetConstGPUValues(), max);
}

void MatrixCUDAMath::MultiplyConstant(Matrix& a, float v)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	MulConstKernel <<<grid, threads >>> (a.GetGPUValues(), v, max);
}

// Misc

void MatrixCUDAMath::FillWith(Matrix& a, float value)
{
	//hipMemset(a.GetGPUValues(), value, a.GetRowCount() * a.GetColumnCount() * sizeof(float));
	unsigned int blockSize = 1;//CalculateMaxBlockSize(a, nullptr, 16);
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	FillKernel <<<grid, threads >>> (a.GetGPUValues(),value, max);
}
