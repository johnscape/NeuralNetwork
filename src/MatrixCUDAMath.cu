#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "NeuralNetwork/MatrixCUDAMath.cuh"

// Kernels

__global__ void MatMulKernel(float* A, float* B, float* C, int m, int n, int k)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0;
	if (col < k && row < m)
	{
		for (int i = 0; i < n; i++)
		{
			sum += A[row * n + i] * B[i * k + col];
		}
		C[row * k + col] = sum;
	}
}

__global__ void MatAddInKernel(float* A, float* B, unsigned int maxNum)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		A[i] += B[i];
}

__global__ void MatAddKerlen(float* A, float* B, float* C, unsigned int maxNum)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		C[i] = A[i] + B[i];
}

__global__ void MatSubInKernel(float* A, float* B, unsigned int maxNum)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		A[i] -= B[i];
}

__global__ void MatSubKernel(float* A, float* B, float* C, unsigned int maxNum)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		C[i] = A[i] - B[i];
}

__global__ void InnerProductKernel(float* A, float* B, unsigned int maxNum)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		A[i] *= B[i];
}

__global__ void FillKernel(float* a, float val, unsigned int maxNum)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		a[i] = val;
}

__global__ void AddConstKernel(float* A, float v, unsigned int maxNum)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		A[i] += v;
}

__global__ void SubConstKernel(float* A, float v, unsigned int maxNum)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		A[i] -= v;
}

__global__ void MulConstKernel(float* A, float v, unsigned int maxNum)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < maxNum)
		A[i] *= v;
}

// Addition
void MatrixCUDAMath::Add(const Matrix& a, const Matrix& b, Matrix& c)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	MatAddKerlen <<<grid, threads >>> (a.GetConstGPUValues(), b.GetConstGPUValues(), c.GetGPUValues(), max);
}


void MatrixCUDAMath::AddIn(Matrix& a, const Matrix& b)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	MatAddInKernel <<<grid, threads >>> (a.GetGPUValues(), b.GetConstGPUValues(), max);
}

void MatrixCUDAMath::AddConstant(Matrix& a, float v)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	AddConstKernel <<<grid, threads >>> (a.GetGPUValues(), v, max);
}

// Subtraction
void MatrixCUDAMath::Subtract(const Matrix& a, const Matrix& b, Matrix& c)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	MatSubKernel <<<grid, threads >>> (a.GetConstGPUValues(), b.GetConstGPUValues(), c.GetGPUValues(), max);
}

void MatrixCUDAMath::SubtractIn(Matrix& a, const Matrix& b)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	MatSubInKernel <<<grid, threads >>> (a.GetGPUValues(), b.GetConstGPUValues(), max);
}

void MatrixCUDAMath::SubtractConstant(Matrix& a, float v)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	SubConstKernel <<<grid, threads >>> (a.GetGPUValues(), v, max);
}

//Multiplication
void MatrixCUDAMath::Multiplication(const Matrix& a, const Matrix& b, Matrix& c)
{
	unsigned int rows = ceil((double)(a.GetRowCount() + BLOCK_SIZE - 1) / (double)BLOCK_SIZE);
	unsigned int cols = ceil((double)(b.GetColumnCount() + BLOCK_SIZE - 1) / (double)BLOCK_SIZE);

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(cols, rows);
	MatMulKernel <<<blocks, threads >>> (a.GetConstGPUValues(), b.GetConstGPUValues(), c.GetGPUValues(), a.GetRowCount(), a.GetColumnCount(), b.GetColumnCount());
}

void MatrixCUDAMath::ElementwiseMultiply(Matrix& a, const Matrix& b)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	InnerProductKernel <<<grid, threads >>> (a.GetGPUValues(), b.GetConstGPUValues(), max);
}

void MatrixCUDAMath::MultiplyConstant(Matrix& a, float v)
{
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	MulConstKernel <<<grid, threads >>> (a.GetGPUValues(), v, max);
}

// Misc

void MatrixCUDAMath::FillWith(Matrix& a, float value)
{
	//hipMemset(a.GetGPUValues(), value, a.GetRowCount() * a.GetColumnCount() * sizeof(float));
	unsigned int blockSize = 1;//CalculateMaxBlockSize(a, nullptr, 16);
	unsigned int max = a.GetColumnCount() * a.GetRowCount();
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(ceil((double)max / (double)threads.x), ceil((double)max / (double)threads.y));
	FillKernel <<<grid, threads >>> (a.GetGPUValues(),value, max);
}
